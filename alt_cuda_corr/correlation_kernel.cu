#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>


#define BLOCK_H 4
#define BLOCK_W 8
#define BLOCK_HW BLOCK_H * BLOCK_W
#define CHANNEL_STRIDE 32


__forceinline__ __device__
bool within_bounds(int h, int w, int H, int W) {
  return h >= 0 && h < H && w >= 0 && w < W;
}

template <typename scalar_t>
__global__ void corr_forward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> fmap1, // 4-dim tensor
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> fmap2, // 4-dim tensor
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> coords,// 5-dim tensor
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> corr,        // 5-dim tensor
    int r) // the output would have (2r + 1)x(2r + 1) neighbors considered. it seems that spatial addressing wrt coords+radius bounds is performed
{
  const int b = blockIdx.x; // current example index in batch
  const int h0 = blockIdx.y * blockDim.x; // blockIdx.y is index of spatial horizontal line divided by blockDim.x (i.e. blockDim.x is loop stride for h0. does it correspond to BLOCK_H?)
  const int w0 = blockIdx.z * blockDim.y; // blockIdx.z is index of spatial vertical line divided by blockDim.y (i.e. blockDim.y is loop stride for w0. does it correspond to BLOCK_W?)
  const int tid = threadIdx.x * blockDim.y + threadIdx.y;

  // fmap1, fmap2 logical dimensions are [B, H, W, C] (fmap1 and fmap2 may well have different spatial dimensions for the spatial pyramid case). do coords have same spatial dims as fmap1? or fmap2?
  const int H1 = fmap1.size(1);
  const int W1 = fmap1.size(2);
  const int H2 = fmap2.size(1);
  const int W2 = fmap2.size(2);
  const int N = coords.size(1);
  const int C = fmap1.size(3);
  
  // it seems that a block accumulates correlations for CHANNEL_STRIDE number of channels (spaced by CHANNEL_STRIDE as well) and BLOCK_HW + 1 in spatial dimensions
  __shared__ scalar_t f1[CHANNEL_STRIDE][BLOCK_HW+1]; // copy buffer for fmap1 features corresponding to the spatial block and strided channels
  __shared__ scalar_t f2[CHANNEL_STRIDE][BLOCK_HW+1]; // copy buffer for fmap1 features corresponding to the spatial block and strided channels
  __shared__ scalar_t x2s[BLOCK_HW]; // x coordinate for spatial location in the block
  __shared__ scalar_t y2s[BLOCK_HW]; // y coordinate for spatial location in the block

  // outer loop over channels?
  for (int c=0; c<C; c+=CHANNEL_STRIDE) { // stride is 32
    
    // fills shared block of feature map f1 from fmap1 (spatial block BLOCK_HW by num of channels CHANNEL_STRIDE)
    for (int k=0; k<BLOCK_HW; k+=BLOCK_HW/CHANNEL_STRIDE) { // here stride is 1
      int k1 = k + tid / CHANNEL_STRIDE;
      int h1 = h0 + k1 / BLOCK_W;
      int w1 = w0 + k1 % BLOCK_W;
      int c1 = tid % CHANNEL_STRIDE;

      auto fptr = fmap1[b][h1][w1];
      if (within_bounds(h1, w1, H1, W1))
        f1[c1][k1] = fptr[c+c1];
      else
        f1[c1][k1] = 0.0; // padded by zero. does padding happen only at the right/bottom ends of tensor because of block granularity?
    }

    __syncthreads(); // wait in all threads and ensure copy-into-f1 completion

    // copy coords into shared block memory, N = 1. this does not depend on channel c. could this be done outside the outer loop?
    for (int n=0; n<N; n++) {
      int h1 = h0 + threadIdx.x; // seems that threadIdx.x is H coordinate within the spatial block
      int w1 = w0 + threadIdx.y; // seems that threadIdx.y is the W coordinate within the spatial block (confusing that "y" ~ W coord)
      if (within_bounds(h1, w1, H1, W1)) {
        x2s[tid] = coords[b][n][h1][w1][0];
        y2s[tid] = coords[b][n][h1][w1][1];
      }

      scalar_t dx = x2s[tid] - floor(x2s[tid]); // fractional part of x2s[tid] ? are coords [0, 1]? or [-1, 1]? or [0, W]?
      scalar_t dy = y2s[tid] - floor(y2s[tid]); // fractional part of y2s[tid] ?

      // iterate over spatial neighboring patches
      int rd = 2*r + 1; // spatial diameter size (i.e. square window side)
      for (int iy=0; iy<rd+1; iy++) {
        for (int ix=0; ix<rd+1; ix++) {
          for (int k=0; k<BLOCK_HW; k+=BLOCK_HW/CHANNEL_STRIDE) {
            int k1 = k + tid / CHANNEL_STRIDE;
            int h2 = static_cast<int>(floor(y2s[k1]))-r+iy;
            int w2 = static_cast<int>(floor(x2s[k1]))-r+ix;
            int c2 = tid % CHANNEL_STRIDE;

            // copy the neighboring patch from fmap2 to f2 (with zero padding)
            auto fptr = fmap2[b][h2][w2];
            if (within_bounds(h2, w2, H2, W2))
              f2[c2][k1] = fptr[c+c2];
            else
              f2[c2][k1] = 0.0;
          }

          __syncthreads();
      
          scalar_t s = 0.0;
          for (int k=0; k<CHANNEL_STRIDE; k++)
            s += f1[k][tid] * f2[k][tid]; // compute the dot product for the spatial and channel block

          int ix_nw = H1*W1*((iy-1) + rd*(ix-1));
          int ix_ne = H1*W1*((iy-1) + rd*ix);
          int ix_sw = H1*W1*(iy + rd*(ix-1));
          int ix_se = H1*W1*(iy + rd*ix);

          scalar_t nw = s * (dy) * (dx);
          scalar_t ne = s * (dy) * (1-dx);
          scalar_t sw = s * (1-dy) * (dx);
          scalar_t se = s * (1-dy) * (1-dx);

          // corr has logical dimensions [B, N, rd*rd, H, W], rd is radius * 2 + 1, N == 1 (defined below in corr_cuda_forward)
          scalar_t* corr_ptr = &corr[b][n][0][h1][w1]; // n and 0 seem exchanged. why? what is corr.stride(1)? does it matter?

          // filling out plane corr[b, 0, 0, h1, w1]. it seems that h1, w1 run over fmap1's spatial dims
          // it seems that a given a (h1, w1) x (h2, w2) dot product is contributed to several target (h1, w1) locations? (proportional to fractional residual)
          if (iy > 0 && ix > 0 && within_bounds(h1, w1, H1, W1))
            *(corr_ptr + ix_nw) += nw;

          if (iy > 0 && ix < rd && within_bounds(h1, w1, H1, W1))
            *(corr_ptr + ix_ne) += ne;

          if (iy < rd && ix > 0 && within_bounds(h1, w1, H1, W1))
            *(corr_ptr + ix_sw) += sw;

          if (iy < rd && ix < rd && within_bounds(h1, w1, H1, W1))
            *(corr_ptr + ix_se) += se;
        }
      } 
    }
  }
}


template <typename scalar_t>
__global__ void corr_backward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> fmap1,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> fmap2,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> coords,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> corr_grad,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> fmap1_grad,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> fmap2_grad,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> coords_grad,
    int r)
{

  const int b = blockIdx.x;
  const int h0 = blockIdx.y * blockDim.x;
  const int w0 = blockIdx.z * blockDim.y;
  const int tid = threadIdx.x * blockDim.y + threadIdx.y;

  const int H1 = fmap1.size(1);
  const int W1 = fmap1.size(2);
  const int H2 = fmap2.size(1);
  const int W2 = fmap2.size(2);
  const int N = coords.size(1);
  const int C = fmap1.size(3);

  __shared__ scalar_t f1[CHANNEL_STRIDE][BLOCK_HW+1];
  __shared__ scalar_t f2[CHANNEL_STRIDE][BLOCK_HW+1];

  __shared__ scalar_t f1_grad[CHANNEL_STRIDE][BLOCK_HW+1];
  __shared__ scalar_t f2_grad[CHANNEL_STRIDE][BLOCK_HW+1];

  __shared__ scalar_t x2s[BLOCK_HW];
  __shared__ scalar_t y2s[BLOCK_HW];

  for (int c=0; c<C; c+=CHANNEL_STRIDE) {

    for (int k=0; k<BLOCK_HW; k+=BLOCK_HW/CHANNEL_STRIDE) {
      int k1 = k + tid / CHANNEL_STRIDE;
      int h1 = h0 + k1 / BLOCK_W;
      int w1 = w0 + k1 % BLOCK_W;
      int c1 = tid % CHANNEL_STRIDE;

      auto fptr = fmap1[b][h1][w1];
      if (within_bounds(h1, w1, H1, W1))
        f1[c1][k1] = fptr[c+c1];
      else
        f1[c1][k1] = 0.0;

      f1_grad[c1][k1] = 0.0;
    }

    __syncthreads();

    int h1 = h0 + threadIdx.x;
    int w1 = w0 + threadIdx.y;

    for (int n=0; n<N; n++) {  
      x2s[tid] = coords[b][n][h1][w1][0];
      y2s[tid] = coords[b][n][h1][w1][1];

      scalar_t dx = x2s[tid] - floor(x2s[tid]);
      scalar_t dy = y2s[tid] - floor(y2s[tid]);

      int rd = 2*r + 1;
      for (int iy=0; iy<rd+1; iy++) {
        for (int ix=0; ix<rd+1; ix++) {
          for (int k=0; k<BLOCK_HW; k+=BLOCK_HW/CHANNEL_STRIDE) {
            int k1 = k + tid / CHANNEL_STRIDE;
            int h2 = static_cast<int>(floor(y2s[k1]))-r+iy;
            int w2 = static_cast<int>(floor(x2s[k1]))-r+ix;
            int c2 = tid % CHANNEL_STRIDE;

            auto fptr = fmap2[b][h2][w2];
            if (within_bounds(h2, w2, H2, W2))
              f2[c2][k1] = fptr[c+c2];
            else
              f2[c2][k1] = 0.0;

            f2_grad[c2][k1] = 0.0;
          }

          __syncthreads();
      
          const scalar_t* grad_ptr = &corr_grad[b][n][0][h1][w1];
          scalar_t g = 0.0;

          int ix_nw = H1*W1*((iy-1) + rd*(ix-1));
          int ix_ne = H1*W1*((iy-1) + rd*ix);
          int ix_sw = H1*W1*(iy + rd*(ix-1));
          int ix_se = H1*W1*(iy + rd*ix);

          if (iy > 0 && ix > 0 && within_bounds(h1, w1, H1, W1))
            g +=  *(grad_ptr + ix_nw) * dy * dx;

          if (iy > 0 && ix < rd && within_bounds(h1, w1, H1, W1))
            g += *(grad_ptr + ix_ne) * dy * (1-dx);

          if (iy < rd && ix > 0 && within_bounds(h1, w1, H1, W1))
            g += *(grad_ptr + ix_sw) * (1-dy) * dx;

          if (iy < rd && ix < rd && within_bounds(h1, w1, H1, W1))
            g += *(grad_ptr + ix_se) * (1-dy) * (1-dx);
            
          for (int k=0; k<CHANNEL_STRIDE; k++) {
            f1_grad[k][tid] += g * f2[k][tid];
            f2_grad[k][tid] += g * f1[k][tid];
          }

          for (int k=0; k<BLOCK_HW; k+=BLOCK_HW/CHANNEL_STRIDE) {
            int k1 = k + tid / CHANNEL_STRIDE;
            int h2 = static_cast<int>(floor(y2s[k1]))-r+iy;
            int w2 = static_cast<int>(floor(x2s[k1]))-r+ix;
            int c2 = tid % CHANNEL_STRIDE;

            scalar_t* fptr = &fmap2_grad[b][h2][w2][0];
            if (within_bounds(h2, w2, H2, W2))
              atomicAdd(fptr+c+c2, f2_grad[c2][k1]);
          }
        }
      } 
    }
    __syncthreads();


    for (int k=0; k<BLOCK_HW; k+=BLOCK_HW/CHANNEL_STRIDE) {
      int k1 = k + tid / CHANNEL_STRIDE;
      int h1 = h0 + k1 / BLOCK_W;
      int w1 = w0 + k1 % BLOCK_W;
      int c1 = tid % CHANNEL_STRIDE;

      scalar_t* fptr = &fmap1_grad[b][h1][w1][0];
      if (within_bounds(h1, w1, H1, W1))
        fptr[c+c1] += f1_grad[c1][k1];
    }
  }
}



std::vector<torch::Tensor> corr_cuda_forward(
  torch::Tensor fmap1,
  torch::Tensor fmap2,
  torch::Tensor coords,
  int radius)
{
  const auto B = coords.size(0);
  const auto N = coords.size(1);
  const auto H = coords.size(2);
  const auto W = coords.size(3);

  const auto rd = 2 * radius + 1;
  auto opts = fmap1.options();
  auto corr = torch::zeros({B, N, rd*rd, H, W}, opts);
  
  const dim3 blocks(B, (H+BLOCK_H-1)/BLOCK_H, (W+BLOCK_W-1)/BLOCK_W);
  const dim3 threads(BLOCK_H, BLOCK_W);

  corr_forward_kernel<float><<<blocks, threads>>>(
    fmap1.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    fmap2.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    coords.packed_accessor32<float,5,torch::RestrictPtrTraits>(),
    corr.packed_accessor32<float,5,torch::RestrictPtrTraits>(),
    radius);

  return {corr};
}

std::vector<torch::Tensor> corr_cuda_backward(
  torch::Tensor fmap1,
  torch::Tensor fmap2,
  torch::Tensor coords,
  torch::Tensor corr_grad,
  int radius)
{
  const auto B = coords.size(0);
  const auto N = coords.size(1);

  const auto H1 = fmap1.size(1);
  const auto W1 = fmap1.size(2);
  const auto H2 = fmap2.size(1);
  const auto W2 = fmap2.size(2);
  const auto C = fmap1.size(3);

  auto opts = fmap1.options();
  auto fmap1_grad = torch::zeros({B, H1, W1, C}, opts);
  auto fmap2_grad = torch::zeros({B, H2, W2, C}, opts);
  auto coords_grad = torch::zeros({B, N, H1, W1, 2}, opts);
    
  const dim3 blocks(B, (H1+BLOCK_H-1)/BLOCK_H, (W1+BLOCK_W-1)/BLOCK_W);
  const dim3 threads(BLOCK_H, BLOCK_W);


  corr_backward_kernel<float><<<blocks, threads>>>(
    fmap1.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    fmap2.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    coords.packed_accessor32<float,5,torch::RestrictPtrTraits>(),
    corr_grad.packed_accessor32<float,5,torch::RestrictPtrTraits>(),
    fmap1_grad.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    fmap2_grad.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    coords_grad.packed_accessor32<float,5,torch::RestrictPtrTraits>(),
    radius);

  return {fmap1_grad, fmap2_grad, coords_grad};
}
